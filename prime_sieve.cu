
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#define idx threadIdx.x
#define COM_COST 20

__global__ void init(bool* prime, int n, int step) {
    int start = step * idx; // including start
    int end = ((start + step) < n) ? (start + step) : n; // excluding end
 
    for(int i = start; i < end; i++) {
        prime[i] = true;
    }
}

__global__ void sieve(bool* prime, int num, int n, int step, int c) {
    // To simulate linear architecture
    for(int i = 0, tmp = 0; i < c * idx * COM_COST; i++) tmp++;

    int start = step * idx; // including start
    int end = ((start + step) < n) ? (start + step) : n; // excluding end
 
    int loc = (start % num) ? start - start % num + num : start; 
    while(loc < end) {
        prime[loc] = false;
        loc += num;
    }
}

// print the maximum prime, for testing
void print_max_prime(bool* prime, int n) {
    for(int i = n - 1; i > n - 100; i--)
        if(prime[i]) {
            printf("The maximum prime less than %d is %d\n", n, i);
            break;
        }
}

int main(int argc, char** argv) {
    int n, p, c;
    if(argc != 4) {
        printf("Usage: ./seive <n> <p> <c> (n > 0,  p > 0 and c >= 0)\n");
        return 1;
    }
    else {
        n = atoi(argv[1]);
        p = atoi(argv[2]);
        c = atoi(argv[3]);
        if(n <= 0 || p <= 0 || c < 0) {
            printf("Usage: ./seive <n> <p> <c> (n > 0,  p > 0 and c >= 0)\n");
            return 1;
        }
    }

    int start_time = clock(); // Start timing

    bool *prime, *d_prime;
    prime = (bool*)malloc(n * sizeof(bool));
    hipMalloc(&d_prime, n * sizeof(bool));

    // Initialize the array in parallel
    int step = (n + p - 1) / p;
    init<<<1, p>>>(d_prime, n, step); // Since my GPU has only one SM

    int sqrt_n = sqrt(n); 
    for(int i = 2; i <= sqrt_n; i++) {
        hipMemcpy(prime + i*sizeof(bool), d_prime + i*sizeof(bool), sizeof(bool), hipMemcpyDeviceToHost);
        if(prime[i])
            sieve<<<1, p>>>(d_prime, i, n, step, c);
    }
    
    hipMemcpy(prime, d_prime, n * sizeof(bool), hipMemcpyDeviceToHost);
    // print_max_prime(prime, n);
    int end_time = clock(); // End timing
    printf("%d\n", (end_time - start_time) / ((int)CLOCKS_PER_SEC / 1000));
    return 0;
}
